#include "hip/hip_runtime.h"
#include "nl-means.h"
#include "shared.cu"

void __global__ nlmeans_kernel(const float3* beauty, int width, int height,
                               float3* denoised)
{
  const int i = blockIdx.x * blockDim.x + threadIdx.x;
  const int j = blockIdx.y * blockDim.y + threadIdx.y;
  if (i >= width || j >= height) return;

  const int K = 8;
  const int P = 7;
  const float sigma_b = 128.0f;
  const float sigma_p = 16.0f;

  const int image_idx = i + width * j;
  const float3 b0 = beauty[image_idx];

  float3 b_sum = make_float3(0.0f);
  float w_sum = 0.0f;

  for (int v = -K; v <= K; ++v) {
    for (int u = -K; u <= K; ++u) {
      float dist = 0.0f;
      for (int t = -P; t <= P; ++t) {
        for (int s = -P; s <= P; ++s) {
          const float3 t0 = beauty[get_image_idx(i + s, j + t, width, height)];
          const float3 t1 =
              beauty[get_image_idx(i + u + s, j + v + t, width, height)];
          dist += length(t0 - t1);
        }
      }
      dist /= (P * P);

      const float3 b1 = beauty[get_image_idx(i + u, j + v, width, height)];

      const float w_b = gaussian_kernel(dist, sigma_b);
      const float w_p = gaussian_kernel(sqrtf(u * u + v * v), sigma_p);
      const float w = w_b * w_p;

      b_sum += w * reinhard(b1);
      w_sum += w;
    }
  }
  w_sum += EPS;

  denoised[image_idx] = reinhard_inverse(b_sum / w_sum);
}

void __host__ nlmeans_kernel_launch(const float3* beauty, const float3* albedo,
                                    const float3* normal, int width, int height,
                                    float3* denoised)
{
  const dim3 threads_per_block(16, 16);
  const dim3 blocks(width / threads_per_block.x + 1,
                    height / threads_per_block.y + 1);
  nlmeans_kernel<<<blocks, threads_per_block>>>(beauty, width, height,
                                                denoised);
}